#include "hip/hip_runtime.h"
/*
	required:
		https://developer.nvidia.com/cuda-downloads
	compile:
		nvcc -shared -o bitonic.dll bitonic.cu
*/

#include <stdlib.h>
#include <stdio.h>

__device__ inline void swap(float &a, float &b)
{
	float temp = a;
	a = b;
	b = temp;
}

__global__ void bitonic_sort_core(float *device_array, int length)
{
	// Gets the if of thread and its index in array:
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	// The number of iterations = log_2(length)
	for (int iteration = 2; iteration <= length; iteration <<= 1) {
		for (int shift = iteration >> 1; shift > 0; shift >>= 1) {
			// The little hack. The shift is always equals 2^n, so there are two situations:
			// 1) i       = xxxx0xxx 
			//    shift   = 00001000
			//    i^shift = xxxx1xxx = i + shift, just like add shift
			//
			// 2) i       = xxxx1xxx 
			//    shift   = 00001000
			//    i^shift = xxxx0xxx = i - shift, i.e. i = j + shift, and then j was processed
			int i_shifted = i ^ shift;
			// i_shifted > i is corresponding to the first case:
			if (i_shifted > i) {
				// Sets the direction of bitonus subsequence:
				// 1) for a first iteration the direction is changing every 2th element
				// 2) for a second iteration the directions is changing every 4th element
				// ...
				if ((i & iteration) != 0) {
					if (device_array[i] < device_array[i_shifted]) {
						swap(device_array[i], device_array[i_shifted]);
					}
				}
				else {
					if (device_array[i] > device_array[i_shifted]) {
						swap(device_array[i], device_array[i_shifted]);
					}
				}
			}
			__syncthreads();
		}
	}
}

extern "C" __declspec ( dllexport ) void bitonic_sort(float *memory_array, int length)
{
	float *device_array;
	size_t size = length * sizeof(float);

	hipMalloc((void**) &device_array, size);
	hipMemcpy(device_array, memory_array, size, hipMemcpyHostToDevice);

	// Calculates the number of threads:
	int count_threads = min(length, 1024);
	int count_blocks = length / count_threads;

	// Inits the dimensions of threads and blocks
	dim3 blocks(count_blocks,1); 
	dim3 threads(count_threads,1);

	// Call the device function
	bitonic_sort_core<<<blocks, threads>>>(device_array, length);

	hipMemcpy(memory_array, device_array, size, hipMemcpyDeviceToHost);
	hipFree(device_array);
}